#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE					// 16, 32, and 64
#define BLOCKSIZE		16			// number of threads per block 
#endif

#ifndef SIZE						// 16K, 32K, 64K, 128K, 256K, and 512K
#define SIZE			16000		// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100			// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

// function prototypes:
float		Ranf(float, float);
int			Ranf(int, int);
void		TimeOfDaySeed();


// Monte Carlo simulation (CUDA Kernel) on the device

__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, int *hits )
{
	// get thread info
	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	// accumulate numHits per thread
	__shared__ float numHits[BLOCKSIZE];
	numHits[tnum] = 0;					

	// randomize the location and radius of the circle:
	float xc = xcs[gid];
	float yc = ycs[gid];
	float  r = rs[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2. * (xc + yc);
	float c = xc * xc + yc * yc - r * r;
	float d = b * b - 4. * a * c;

	// case A: circle completely missed (d < 0.)
	if (d < 0.)
	{
		numHits[tnum] = 0;
		goto finish_analysis;
	}
		
	// if not case A, hits the circle:
	// get the first intersection:
	d = sqrtf(d);
	float t1 = (-b + d) / (2. * a);						 // time to intersect the circle
	float t2 = (-b - d) / (2. * a);						 // time to intersect the circle
	float tmin = t1 < t2 ? t1 : t2;					     // only care about the first intersection

	// case B: circle engulfs line (tmin < 0.)
	if (tmin < 0.)
	{
		numHits[tnum] = 0;
		goto finish_analysis;
	}
		
	// if not case A or case B, where does it intersect the circle?
	float xcir = tmin;
	float ycir = tmin;

	// get the unitized normal vector at the point of intersection:
	float nx = xcir - xc;
	float ny = ycir - yc;
	float n = sqrtf(nx * nx + ny * ny);
	nx /= n;											// unit vector
	ny /= n;											// unit vector
							
	// get the unitized incoming vector:
	float inx = xcir - 0.;
	float iny = ycir - 0.;
	float in = sqrtf(inx * inx + iny * iny);
	inx /= in;											// unit vector
	iny /= in;											// unit vector

	// get the outgoing (bounced) vector:
	float dot = inx * nx + iny * ny;
	float outx = inx - 2. * nx * dot;					// angle of reflection = angle of incidence`
	float outy = iny - 2. * ny * dot;					// angle of reflection = angle of incidence`

	// find out if it hits the infinite plate:
	float t = (0. - ycir) / outy;

	// case C: line bounced back up (t < 0.)
	if (t < 0.)
	{
		numHits[tnum] = 0;
		goto finish_analysis;
	}

	// case D: if not case A, B, or C, line hit the plate
	numHits[tnum] = 1;

	// do the reduction (all threads execute simultaneously)
	finish_analysis:									
	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			numHits[tnum] += numHits[tnum + offset];
		}
	}

	// add to results array after all threads have finished
	__syncthreads();						
	if (tnum == 0)
		hits[wgNum] = numHits[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	TimeOfDaySeed();

	// allocate host memory:

	float *hXCS    = new float[ SIZE ];			// x centers
	float *hYCS    = new float[ SIZE ];			// y centers
	float *hRS     = new float[ SIZE ];			// radius
	int   *hHits   = new int  [ SIZE ];			// return results from work groups to add on CPU

	for (int n = 0; n < SIZE; n++)
	{
		hXCS[n]    = Ranf(XCMIN, XCMAX);
		hYCS[n]    = Ranf(YCMIN, YCMAX);
		hRS[n]     = Ranf(RMIN, RMAX);
		hHits[n]   = 0;
	}

	// allocate device memory:

	float *dXCS, *dYCS, *dRS;
	int   *dHits;

	dim3 dimsXCS(SIZE, 1, 1);
	dim3 dimsYCS(SIZE, 1, 1);
	dim3 dimsRS(SIZE, 1, 1);
	dim3 dimsHits(SIZE / BLOCKSIZE, 1, 1);

	//__shared__ float prods[SIZE/BLOCKSIZE];

	hipError_t status;			
	status = hipMalloc(reinterpret_cast<void **>(&dXCS), SIZE * sizeof(float));
		checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void **>(&dYCS), SIZE * sizeof(float));
		checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void **>(&dRS),  SIZE * sizeof(float));
		checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void **>(&dHits), (SIZE / BLOCKSIZE) * sizeof(int));
		checkCudaErrors(status);


	// copy host memory to the device:

	status = hipMemcpy(dXCS, hXCS, SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);
	status = hipMemcpy(dYCS, hYCS, SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);
	status = hipMemcpy(dRS,  hRS,  SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for (int t = 0; t < NUMTRIALS; t++)
	{
			MonteCarlo << < grid, threads >> > (dXCS, dYCS, dRS, dHits);
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );
		
	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "Block Size = %10d, Array Size = %10d, MegaTrials/Second = %10.2lf\n", BLOCKSIZE, SIZE, megaTrialsPerSecond );
	
	// copy result from the device to the host:

	status = hipMemcpy( hHits, dHits, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the probability -- should be about 42%? 

	float totalHits = 0.;
	for (int i = 0; i < SIZE/BLOCKSIZE; i++)						
	{
		totalHits += hHits[i];
	}
	float probability = (totalHits / SIZE) * 100;
	printf("Probability: %10.2lf\n", probability);

	// read performance and probability to file

	FILE* outputResults = fopen("monteCarlo_results.csv", "a");
	if (outputResults == NULL)
	{
		printf("Error: no output file\n");
		exit(1);
	}

	fprintf(outputResults, "%d, %d, %.2lf, %.2lf\n", BLOCKSIZE, SIZE, megaTrialsPerSecond, probability);

	fclose(outputResults);

	// clean up memory:

	delete [ ] hXCS;
	delete [ ] hYCS;
	delete [ ] hRS;
	delete [ ] hHits;

	status = hipFree(dXCS);
		checkCudaErrors(status);
	status = hipFree(dYCS);
		checkCudaErrors(status);
	status = hipFree(dRS);
		checkCudaErrors(status);


	return 0;
}

// functions for random numbers
float
Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}

// seed time of day
void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000.*seconds);    // milliseconds
	srand(seed);
}